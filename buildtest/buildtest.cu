#include "hip/hip_runtime.h"
#include "buildtest.hpp"

#include <cstdio>
#include <iostream>

namespace BuildTest {
  __global__ void computeIndex(int* arr, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
      arr[i] = i;
    }
  }

  #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
  inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
  {
     if (code != hipSuccess) 
     {
        fprintf(stderr,"GPUassert: \"%s\", at %s, line %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
     }
  }

  void printCudaInfo()
  {
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    gpuErrchk(hipGetDeviceCount(&deviceCount));

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
      hipDeviceProp_t deviceProps;
      gpuErrchk(hipGetDeviceProperties(&deviceProps, i));
      printf("Device %d: %s\n", i, deviceProps.name);
      printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
      printf("   Global mem: %.0f MB\n",
          static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
      printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
  }

  int runBuildTest() {
    printCudaInfo();

    std::cout << "Running the tests..." << std::endl;

    const int N = 1 << 12;
    const int THREADS_PER_BLK = 64;

    int* arr;
    gpuErrchk(hipMalloc((void**)&arr, N * sizeof(int)));

    computeIndex<<<N/THREADS_PER_BLK, THREADS_PER_BLK>>>(arr, N);

    int localArr[N];

    gpuErrchk(hipMemcpy(localArr, arr, N * sizeof(int), hipMemcpyDeviceToHost));

    int numErrors = 0;
    for (int i = 0; i < N; i++) {
      if (i != localArr[i]) {
        numErrors++;
      }
    }

    if (numErrors == 0) {
      std::cout << "+++ CUDA build test passed! +++" << std::endl;
    } else {
      std::cerr << "--- FAILED CUDA BUILD TEST ---" << std::endl;
    }

    return numErrors;
  }
}
